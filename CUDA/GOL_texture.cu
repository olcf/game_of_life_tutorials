#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
 
#define SRAND_VALUE 1985
 
texture<int,2> gridTex;
 
__global__ void GOL(int dim, int *newGrid)
{
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int id = iy * dim + ix;
 
    int numNeighbors;
 
    float iyTex = (iy + 0.5f)/dim;
    float ixTex = (ix + 0.5f)/dim;
    float oneTex = 1.0f/dim;
 
    if(iy < dim && ix < dim)
{
    //Get the number of neighbors for a given grid point
    numNeighbors = tex2D(gridTex, ixTex+oneTex, iyTex) //right/left
                 + tex2D(gridTex, ixTex-oneTex, iyTex)
                 + tex2D(gridTex, ixTex, iyTex+oneTex) //upper/lower
                 + tex2D(gridTex, ixTex, iyTex-oneTex)
                 + tex2D(gridTex, ixTex-oneTex, iyTex-oneTex) //diagonals
                 + tex2D(gridTex, ixTex-oneTex, iyTex+oneTex)
                 + tex2D(gridTex, ixTex+oneTex, iyTex-oneTex)
                 + tex2D(gridTex, ixTex+oneTex, iyTex+oneTex);

    int cell = tex2D(gridTex, ixTex, iyTex);

    //Here we have explicitly all of the game rules
    if (cell == 1 && numNeighbors < 2)
        newGrid[id] = 0;
    else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
        newGrid[id] = 1;
    else if (cell == 1 && numNeighbors > 3)
        newGrid[id] = 0;
    else if (cell == 0 && numNeighbors == 3)
         newGrid[id] = 1;
    else
       newGrid[id] = cell;
 
}
}
 
int main(int argc, char* argv[])
{
    int i,j,iter;
    int* h_grid; //Grid on host
    hipArray* d_grid; //Grid on device
    int* d_newGrid; //Second grid used on device only
 
    int dim = 1024; //Linear dimension of our grid - not counting ghost cells
    int maxIter = 1<<10; //Number of game steps
 
    size_t bytes = sizeof(int)*dim*dim;
    //Allocate host Grid used for initial setup and read back from device
    h_grid = (int*)malloc(bytes);
 
    //Allocate device grids
    hipMallocArray(&d_grid, &gridTex.channelDesc, dim, dim);
    hipMalloc(&d_newGrid, bytes);
 
    //Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 0; i<dim; i++) {
        for(j = 0; j<dim; j++) {
            h_grid[i*dim+j] = rand() % 2;
        }
    }
 
    //Copy over initial game grid (Dim-1 threads)
    hipMemcpyToArray (d_grid, 0, 0, h_grid, bytes, hipMemcpyHostToDevice);
    hipBindTextureToArray(gridTex, d_grid);
 
    gridTex.normalized = true;
    gridTex.addressMode[0] = hipAddressModeBorder;
    gridTex.addressMode[1] = hipAddressModeBorder;
 
    dim3 dimBlock(8,8);
    int linGrid = (int)ceil(dim/(float)dimBlock.x);
    dim3 dimGrid(linGrid,linGrid);
 
    //Main game loop
    for (iter = 0; iter<maxIter; iter++) {
        GOL<<<dimGrid,dimBlock>>>(dim, d_newGrid);
 
        //Swap our grids and iterate again
        hipMemcpyToArray (d_grid, 0, 0, d_newGrid, bytes, hipMemcpyDeviceToDevice);
    }//iter loop
 
    //Copy back results and sum
    hipMemcpy(h_grid, d_newGrid, bytes, hipMemcpyDeviceToHost);
 
    //Sum up alive cells and print results
    int total = 0;
    for (i = 0; i<dim; i++) {
        for (j = 0; j<dim; j++) {
            total += h_grid[i*dim+j];
        }
    }
    printf("Total Alive: %d\n", total);
 
    hipFree(d_grid);
    hipFree(d_newGrid);
    free(h_grid);
 
    return 0;
}
